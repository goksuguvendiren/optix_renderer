#include "hip/hip_runtime.h"
//
// Created by Goksu Guvendiren on 02.20.2019
//

#include <optixu/optixu_math_namespace.h>
#include "../../include/light_sources/point.hpp"
#include "../../include/light_sources/area.hpp"
#include "random.h"

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtBuffer<grpt::area_light>       lights;
rtBuffer<grpt::point_light>      point_lights;


RT_PROGRAM void diffuse()
{
    optix::float3 world_geo_normal   = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    optix::float3 world_shade_normal = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    optix::float3 ffnormal           = optix::faceforward( world_shade_normal, -ray.direction, world_geo_normal );

    float3 Ka = optix::make_float3( 0.3f, 0.3f, 0.3f );
    float3 Kd = optix::make_float3( 0.6f, 0.7f, 0.8f );
    float3 ambient_color = optix::make_float3( 0.31f, 0.33f, 0.28f );

//    box_matl["Ka"]->setFloat( 0.3f, 0.3f, 0.3f );
//    box_matl["Kd"]->setFloat( 0.6f, 0.7f, 0.8f );
//    box_matl["Ks"]->setFloat( 0.8f, 0.9f, 0.8f );
//    box_matl["phong_exp"]->setFloat( 88 );

    //ambient
    optix::float3 result = Ka * ambient_color * diffuse_color;
    optix::float3 hit_point = ray.origin + t_hit * ray.direction;

    for (int i = 0; i < point_lights.size(); ++i)
    {
        grpt::point_light light = point_lights[i];

        optix::float3 L = optix::normalize(light.Position() - hit_point);
        float cos_theta = optix::dot(L, ffnormal);

        if (cos_theta > 0)
        {
            result += Ka * cos_theta * light.Emission();
        }
    }

    current_prd.result = result;
}


RT_PROGRAM void diffuse_path()
{
    float3 world_shading_normal   = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = optix::faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    optix::cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        grpt::area_light light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = optix::length(light_pos - hitpoint);
        const float3 L     = optix::normalize(light_pos - hitpoint);
        const float  nDl   = optix::dot( ffnormal, L );
        const float  LnDl  = optix::dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            optix::Ray shadow_ray = optix::make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = optix::length(optix::cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    num_lights = point_lights.size();
    for(int i = 0; i < num_lights; ++i)
    {
        grpt::point_light light = point_lights[i];
        const float3 light_pos = light.Position();

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist    = optix::length(light_pos - hitpoint);
        const float3 L        = optix::normalize(light_pos - hitpoint);
        const float  costheta = optix::dot( ffnormal, L );

        // cast shadow ray
        if ( costheta > 0.0f)
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            optix::Ray shadow_ray = optix::make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
//                const float A = 1;//optix::length(optix::cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                optix::float3 color = light.Emission() * (diffuse_color);
                result += color;
            }
//            else
//                result += optix::make_float3(0.8);
        }
    }

    current_prd.radiance = result;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}

