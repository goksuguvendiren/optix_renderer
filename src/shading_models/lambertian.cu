#include "hip/hip_runtime.h"
//
// Created by Goksu Guvendiren on 02.20.2019
//

#include <optixu/optixu_math_namespace.h>
#include "../../include/light_sources/point.hpp"
#include "../../include/light_sources/area.hpp"
#include "random.h"

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtBuffer<grpt::area_light>       lights;
rtBuffer<grpt::point_light>      point_lights;

RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = optix::faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    optix::cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        grpt::area_light light = lights[i];
        const float z1 = rnd(current_prd.seed);
        const float z2 = rnd(current_prd.seed);
        const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = optix::length(light_pos - hitpoint);
        const float3 L     = optix::normalize(light_pos - hitpoint);
        const float  nDl   = optix::dot( ffnormal, L );
        const float  LnDl  = optix::dot( light.normal, L );

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            optix::Ray shadow_ray = optix::make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float A = optix::length(optix::cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
                result += light.emission * weight;
            }
        }
    }

    num_lights = point_lights.size();
    for(int i = 0; i < num_lights; ++i)
    {
        grpt::point_light light = point_lights[i];
        const float3 light_pos = light.Position();

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist    = optix::length(light_pos - hitpoint);
        const float3 L        = optix::normalize(light_pos - hitpoint);
        const float  costheta = optix::dot( ffnormal, L );

        // cast shadow ray
        if ( costheta > 0.0f)
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            optix::Ray shadow_ray = optix::make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
//                const float A = 1;//optix::length(optix::cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                optix::float3 color = light.Emission() * (diffuse_color);
                result += color;
            }
//            else
//                result += optix::make_float3(0.8);
        }
    }

    current_prd.radiance = result;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}

