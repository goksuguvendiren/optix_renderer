#include "hip/hip_runtime.h"
//
// Created by Goksu Guvendiren on 02.20.2019
//

#include <optixu/optixu_math_namespace.h>
#include "../../optixPathTracer.h"
#include "include/light_sources/point.hpp"
#include "random.h"

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtBuffer<ParallelogramLight>     area_lights;
rtBuffer<grpt::point_light>      point_lights;

RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = optix::faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    optix::cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    current_prd.attenuation = current_prd.attenuation * diffuse_color;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = area_lights.size();
    float3 result = make_float3(0.0f);

    num_lights = point_lights.size();
    for(int i = 0; i < num_lights; ++i)
    {
        grpt::point_light light = point_lights[i];
        const float3 light_pos = light.Position();

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist    = optix::length(light_pos - hitpoint);
        const float3 L        = optix::normalize(light_pos - hitpoint);
        const float  costheta = optix::dot( ffnormal, L );

        // cast shadow ray
        if ( costheta > 0.0f)
        {
//            result = optix::make_float3(0.3);
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            optix::Ray shadow_ray = optix::make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
//                const float A = 1;//optix::length(optix::cross(light.v1, light.v2));
                // convert area based pdf to solid angle
                optix::float3 color = optix::make_float3(0.3);//light.Emission();
                result += color;
            }
//            else
//                result += optix::make_float3(0.8);
        }
    }

    current_prd.radiance = result;
}

//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}

