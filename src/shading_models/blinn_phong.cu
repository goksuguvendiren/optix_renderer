#include "hip/hip_runtime.h"
//
// Created by Göksu Güvendiren on 2019-03-01.
//

#include <optixu/optixu_math_namespace.h>
#include "../../optixPathTracer.h"
#include "../../include/light_sources/point.hpp"

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );
rtDeclareVariable(PerRayData_pathtrace_shadow, prd_shadow, rtPayload, );

rtBuffer<grpt::point_light>      point_lights;

RT_PROGRAM void closest_hit()
{
    optix::float3 world_geo_normal   = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    optix::float3 world_shade_normal = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    optix::float3 ffnormal           = optix::faceforward( world_shade_normal, -ray.direction, world_geo_normal );

    float3 Ka = optix::make_float3( 0.3f, 0.3f, 0.3f );
    float3 Kd = optix::make_float3( 0.6f, 0.7f, 0.8f );
    float3 Ks = optix::make_float3( 0.8f, 0.9f, 0.8f );
    float  phong_exp = 88;
    float3 ambient_color = optix::make_float3( 0.31f, 0.33f, 0.28f );

    //ambient
    optix::float3 result = Ka * ambient_color * diffuse_color;
    optix::float3 hit_point = ray.origin + t_hit * ray.direction;

    for (int i = 0; i < point_lights.size(); ++i)
    {
        grpt::point_light light = point_lights[i];

        PerRayData_pathtrace_shadow shadow_payload;2
        shadow_payload.inShadow = false;

        float light_distance = optix::length(light.Position() - hit_point);
        optix::float3 L = optix::normalize(light.Position() - hit_point);
        optix::float3 ray_pos = hit_point + L * scene_epsilon;

        optix::Ray shadow_ray = optix::make_Ray(ray_pos, L, 1, scene_epsilon, light_distance);
        rtTrace(top_object, shadow_ray, shadow_payload);

        if (shadow_payload.inShadow) continue;

        float cos_theta = optix::dot(L, ffnormal);

        if (cos_theta > 0)
        {
            // diffuse term
            result += Ka * cos_theta * light.Emission();

            // specular term
            optix::float3 H = optix::normalize(L - ray.direction);
            float cos_alpha = optix::dot(H, ffnormal);
            if (cos_alpha > 0)
            {
                result += Ks * light.Emission() * pow(cos_alpha, phong_exp);
            }
        }
    }

    current_prd.result = result;
}

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void any_hit()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}