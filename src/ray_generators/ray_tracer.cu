#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "../../include/light_sources/area.hpp"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};
// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );

rtBuffer<float4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
    size_t2 screen = output_buffer.size();

    float2 dir = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = normalize(dir.x * U + dir.y * V + W);

    float3 result = make_float3(0.0f);

    // Initialze per-ray data
    PerRayData_pathtrace prd;
    prd.result = make_float3(0.f);
    prd.attenuation = make_float3(1.f);
    prd.countEmitted = true;
    prd.done = false;
    prd.depth = 0;

    Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_float4(prd.result, 1.0f);
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.result = bg_color;
    current_prd.done = true;
}


